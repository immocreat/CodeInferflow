#include "hip/hip_runtime.h"
#include "cuda_util.h"
#include "sslib/string.h"
#include "sslib/log.h"
#include "tools/profiler.h"

INFER_FLOW_BEGIN

using namespace sslib;

//static
bool CudaUtil::CheckReturnCode(hipError_t ret_code, const char *error_title)
{
    if (ret_code != hipSuccess)
    {
        if (String::IsNullOrEmpty(error_title))
        {
            LogError("Failed to call a cuda function: %d (%s)",
                ret_code, hipGetErrorString(ret_code));
        }
        else
        {
            LogError("%s: %d (%s)", error_title, ret_code,
                hipGetErrorString(ret_code));
        }
        return false;
    }

    return true;
}

//static
int CudaUtil::DeviceCount()
{
    int count = 0;
    hipGetDeviceCount(&count);
    return count;
}

//static
int CudaUtil::GetDevice()
{
    int device_id = 0;
    hipGetDevice(&device_id);
    return device_id;
}

//static
void CudaUtil::SetDevice(int device_id)
{
    hipSetDevice(device_id);
}

bool CudaUtil::DeviceSynchronize(const char *error_title)
{
    hipDeviceSynchronize(); //Wait for kernels to finish

    auto status = hipGetLastError();
    bool ret = CheckReturnCode(status, error_title);
    return ret;
}

//static
int CudaUtil::GetDeviceMemoryInfo(DeviceMemoryInfo &mem_info, int device_id)
{
    int device_count = DeviceCount();
    int current_device_id = GetDevice();
    if (device_id < 0 || device_id >= device_count) {
        device_id = current_device_id;
    }

    if (device_id != current_device_id) {
        SetDevice(device_id);
    }

    size_t free_bytes = 0, total_bytes = 0;
    hipMemGetInfo(&free_bytes, &total_bytes);

    mem_info.free_bytes = free_bytes;
    mem_info.total_bytes = total_bytes;
    mem_info.used_bytes = total_bytes - free_bytes;
    mem_info.free_gb = free_bytes / 1024.0f / 1024 / 1024;
    mem_info.total_gb = total_bytes / 1024.0f / 1024 / 1024;
    mem_info.used_gb = mem_info.total_gb - mem_info.free_gb;

    if (device_id != current_device_id) {
        SetDevice(current_device_id);
    }
    return device_id;
}

//static
int CudaUtil::LogDeviceMemoryInfo(int device_id)
{
    DeviceMemoryInfo mi;
    int ret_device_id = GetDeviceMemoryInfo(mi, device_id);
    LogKeyInfo("device %d: total %.3f GB, used %.3f GB, free %.3f GB",
        ret_device_id, mi.total_gb, mi.used_gb, mi.free_gb);

    return ret_device_id;
}

//static
bool CudaUtil::DeviceToDeviceMemcpy(void *dst, const void *src, size_t byte_count)
{
    auto ret_code = hipMemcpy(dst, src, byte_count, hipMemcpyDeviceToDevice);
    if (ret_code != hipSuccess)
    {
        LogError("Failed to copy %d byte(s) from device to device: %d (%s)",
            byte_count, ret_code, hipGetErrorString(ret_code));
        return false;
    }

    return true;
}

//static
bool CudaUtil::DeviceToHostMemcpy(void *dst, const void *src, size_t byte_count)
{
    auto ret_code = hipMemcpy(dst, src, byte_count, hipMemcpyDeviceToHost);
    if (ret_code != hipSuccess)
    {
        LogError("Failed to copy %d byte(s) to host: %d (%s)",
            byte_count, ret_code, hipGetErrorString(ret_code));
        return false;
    }

    return true;
}

//static
bool CudaUtil::HostToDeviceMemcpy(void *dst, const void *src, size_t byte_count)
{
    TIMEIT()
    auto ret_code = hipMemcpy(dst, src, byte_count, hipMemcpyHostToDevice);
    if (ret_code != hipSuccess)
    {
        LogError("Failed to copy %d byte(s) from host: %d (%s)",
            byte_count, ret_code, hipGetErrorString(ret_code));
        return false;
    }

    return true;
}

//static
bool CudaUtil::MemcpyPeer(void *dst, int dst_device, const void *src,
    int src_device, size_t byte_count)
{
    if (dst_device == src_device) {
        return DeviceToDeviceMemcpy(dst, src, byte_count);
    }

    auto ret_code = hipMemcpyPeer(dst, dst_device, src, src_device, byte_count);
    if (ret_code != hipSuccess)
    {
        LogError("Failed to call hipMemcpyPeer: %d (%s)",
            ret_code, hipGetErrorString(ret_code));
        return false;
    }

    bool ret = DeviceSynchronize();
    return ret;
}

INFER_FLOW_END

