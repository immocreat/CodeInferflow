#include "hip/hip_runtime.h"
#include "cublas_engine.h"
#include "sslib/log.h"
#include <hip/hip_runtime_api.h>

INFER_FLOW_BEGIN

using namespace sslib;

CublasEngine::~CublasEngine()
{
    Clear();
}

void CublasEngine::Clear()
{
    hipblasDestroy(handle_);
}

bool CublasEngine::Init()
{
    auto ret_code = hipblasCreate(&handle_);
    if (ret_code != HIPBLAS_STATUS_SUCCESS) {
        LogError("cuBLAS error %d in calling hipblasCreate", ret_code);
        return false;
    }

    //ret_code = hipblasSetMathMode(handle_, HIPBLAS_TF32_TENSOR_OP_MATH);
    ret_code = hipblasSetMathMode(handle_, HIPBLAS_DEFAULT_MATH);
    //ret_code = hipblasSetMathMode(handle_, HIPBLAS_TENSOR_OP_MATH);
    if (ret_code != HIPBLAS_STATUS_SUCCESS) {
        LogError("cuBLAS error %d in calling hipblasSetMathMode", ret_code);
        return false;
    }

    return true;
}

bool CublasEngine::GemmEx(DeviceTensor &C, const DeviceTensor &A, const DeviceTensor &B,
    float alpha, float beta, bool is_b_column_major, bool is_comp32, int alg_id)
{
    bool ret = true;
    int m = A.Rows(), k = A.Columns();
    int n = is_b_column_major ? B.Rows() : B.Columns();

    bool is_fp16 = A.data_type == ElementType::F16 && B.data_type == ElementType::F16
        && C.data_type == ElementType::F16;
    bool is_fp32 = A.data_type == ElementType::F32 && B.data_type == ElementType::F32
        && C.data_type == ElementType::F32;

    //hipDeviceSynchronize();
    //hipProfilerStart();

    if (is_fp16)
    {
        const half *a_data = A.data_f16();
        const half *b_data = B.data_f16();
        half *c_data = C.data_f16();
        ret = GemmEx_F16(m, n, k, a_data, b_data, c_data, alpha, beta,
            is_b_column_major, is_comp32, alg_id);
    }
    else if (is_fp32)
    {
        const float *a_data = A.data_f32();
        const float *b_data = B.data_f32();
        float *c_data = C.data_f32();
        ret = GemmEx_F32(m, n, k, a_data, b_data, c_data, alpha, beta,
            is_b_column_major, alg_id);
    }
    else
    {
        LogError("Not implemented yet (should both be fp16 or fp32)!");
        ret = false;
    }

    //hipProfilerStop();
    return ret;
}

bool CublasEngine::GemmExBatch(DeviceTensor &C, const DeviceTensor &A, const DeviceTensor &B,
    DeviceMemoryHeap &heap, float alpha, float beta, bool is_b_column_major, int alg_id)
{
    bool ret = true;
    if (is_b_column_major)
    {
        if (A.ne[2] != B.ne[2] || A.ne[0] != B.ne[0])
        {
            LogError("A and B are not compatible: %d vs. %d and %d vs. %d",
                A.ne[2], B.ne[2], A.ne[0], B.ne[0]);
            return false;
        }

        if (A.ne[2] != C.ne[2] || C.ne[1] != A.ne[1] || C.ne[0] != B.ne[1])
        {
            LogError("C (%d, %d, %d) is not compatible with A (%d, %d, %d) and B (%d, %d, %d).",
                C.ne[0], C.ne[1], C.ne[2], A.ne[0], A.ne[1], A.ne[2], B.ne[0], B.ne[1], B.ne[2]);
            return false;
        }
    }
    else
    {
        if (A.ne[2] != B.ne[2] || A.ne[0] != B.ne[1])
        {
            LogError("A and B are not compatible: %d vs. %d and %d vs. %d",
                A.ne[2], B.ne[2], A.ne[0], B.ne[1]);
            return false;
        }

        if (A.ne[2] != C.ne[2] || C.ne[1] != A.ne[1] || C.ne[0] != B.ne[0])
        {
            LogError("C (%d, %d, %d) is not compatible with A (%d, %d, %d) and B (%d, %d, %d).",
                C.ne[0], C.ne[1], C.ne[2], A.ne[0], A.ne[1], A.ne[2], B.ne[0], B.ne[1], B.ne[2]);
            return false;
        }
    }

    int batch_size = A.ne[2];
    int m = A.ne[1], k = A.ne[0];
    int n = is_b_column_major ? B.ne[1] : B.ne[0];

    bool is_fp16 = A.data_type == ElementType::F16 && B.data_type == ElementType::F16
        && C.data_type == ElementType::F16;
    bool is_fp32 = A.data_type == ElementType::F32 && B.data_type == ElementType::F32
        && C.data_type == ElementType::F32;

    if (is_fp16)
    {
        vector<const half*> ha(batch_size), hb(batch_size);
        vector<half*> hc(batch_size);
        for (int depth = 0; depth < batch_size; depth++)
        {
            ha[depth] = (const half*)A.RowData(0, depth);
            hb[depth] = (const half*)B.RowData(0, depth);
            hc[depth] = (half*)C.RowData(0, depth);
        }

        int bytes = sizeof(half*) * batch_size;
        half **da = (half**)heap.New(bytes);
        half **db = (half**)heap.New(bytes);
        half **dc = (half**)heap.New(bytes);
        CudaUtil::HostToDeviceMemcpy(da, ha.data(), bytes);
        CudaUtil::HostToDeviceMemcpy(db, hb.data(), bytes);
        CudaUtil::HostToDeviceMemcpy(dc, hc.data(), bytes);

        bool is_comp32 = false;
        ret = GemmExBatch_F16(m, n, k, (const half**)da, (const half**)db, dc,
            alpha, beta, is_b_column_major, is_comp32, batch_size, alg_id);
    }
    else if (is_fp32)
    {
        vector<const float*> ha(batch_size), hb(batch_size);
        vector<float*> hc(batch_size);
        for (int depth = 0; depth < batch_size; depth++)
        {
            ha[depth] = (const float*)A.RowData(0, depth);
            hb[depth] = (const float*)B.RowData(0, depth);
            hc[depth] = (float*)C.RowData(0, depth);
        }

        int bytes = sizeof(float*) * batch_size;
        float **da = (float**)heap.New(bytes);
        float **db = (float**)heap.New(bytes);
        float **dc = (float**)heap.New(bytes);
        CudaUtil::HostToDeviceMemcpy(da, ha.data(), bytes);
        CudaUtil::HostToDeviceMemcpy(db, hb.data(), bytes);
        CudaUtil::HostToDeviceMemcpy(dc, hc.data(), bytes);

        ret = GemmExBatch_F32(m, n, k, (const float**)da, (const float**)db, dc,
            alpha, beta, is_b_column_major, batch_size, alg_id);
    }
    else
    {
        LogError("Not implemented yet (should both be fp16 or fp32)!");
        ret = false;
    }

    return ret;
}

bool CublasEngine::GemmEx_F16(int m, int n, int k, const half *A, const half *B, half *C,
    float alpha, float beta, bool is_b_column_major, bool is_comp32,
    int alg_id)
{
    hipblasStatus_t ret_code = HIPBLAS_STATUS_SUCCESS;
    if (is_b_column_major)
    {
        if (is_comp32)
        {
            ret_code = GemmEx_Inner_FP16_Comp32(HIPBLAS_OP_T, HIPBLAS_OP_N,
                m, n, k,
                A, k, B, k, C, m,
                &alpha, &beta, alg_id);
            //ret_code = GemmEx_Inner_FP16_Comp32(HIPBLAS_OP_T, HIPBLAS_OP_N,
            //    n, m, k,
            //    B, n, A, k, C, n,
            //    &alpha, &beta, alg_id);
        }
        else
        {
            ret_code = GemmEx_Inner_FP16_Comp16(HIPBLAS_OP_T, HIPBLAS_OP_N,
                m, n, k,
                A, k, B, k, C, m,
                &alpha, &beta, alg_id);
            //ret_code = GemmEx_Inner_FP16_Comp16(HIPBLAS_OP_N, HIPBLAS_OP_T,
            //    n, m, k,
            //    B, n, A, m, C, n,
            //    &alpha, &beta, alg_id);
        }
    }
    else
    {
        if (is_comp32)
        {
            ret_code = GemmEx_Inner_FP16_Comp32(HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, m, k,
                B, n, A, k, C, n,
                &alpha, &beta, alg_id);
        }
        else
        {
            ret_code = GemmEx_Inner_FP16_Comp16(HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, m, k,
                B, n, A, k, C, n,
                &alpha, &beta, alg_id);
        }
    }

    hipDeviceSynchronize();

    if (ret_code != HIPBLAS_STATUS_SUCCESS) {
        LogError("cuBLAS error %d in calling hipblasGemmEx", ret_code);
        return false;
    }
    return true;
}

bool CublasEngine::GemmExBatch_F16(int m, int n, int k, const half **array_a,
    const half **array_b, half **array_c, float alpha, float beta,
    bool is_b_column_major, bool is_comp32, int batch_num, int alg_id)
{
    hipblasStatus_t ret_code = HIPBLAS_STATUS_SUCCESS;
    if (is_b_column_major)
    {
        if (is_comp32)
        {
            ret_code = GemmExBatch_Inner_FP16_Comp32(HIPBLAS_OP_T, HIPBLAS_OP_N,
                m, n, k,
                array_a, k, array_b, k, array_c, m,
                &alpha, &beta, batch_num, alg_id);
        }
        else
        {
            ret_code = GemmExBatch_Inner_FP16_Comp16(HIPBLAS_OP_T, HIPBLAS_OP_N,
                m, n, k,
                array_a, k, array_b, k, array_c, m,
                &alpha, &beta, batch_num, alg_id);
        }
    }
    else
    {
        if (is_comp32)
        {
            ret_code = GemmExBatch_Inner_FP16_Comp32(HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, m, k,
                array_b, n, array_a, k, array_c, n,
                &alpha, &beta, batch_num, alg_id);
        }
        else
        {
            ret_code = GemmExBatch_Inner_FP16_Comp16(HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, m, k,
                array_b, n, array_a, k, array_c, n,
                &alpha, &beta, batch_num, alg_id);
        }
    }

    hipDeviceSynchronize();

    if (ret_code != HIPBLAS_STATUS_SUCCESS) {
        LogError("cuBLAS error %d in calling hipblasGemmBatchedEx", ret_code);
        return false;
    }
    return true;
}

bool CublasEngine::GemmExBatch_F32(int m, int n, int k, const float **array_a,
    const float **array_b, float **array_c, float alpha, float beta,
    bool is_b_column_major, int batch_size, int alg_id)
{
    hipblasStatus_t ret_code = HIPBLAS_STATUS_SUCCESS;
    if (is_b_column_major)
    {
        ret_code = GemmExBatch_Inner_FP32(HIPBLAS_OP_T, HIPBLAS_OP_N,
            m, n, k,
            array_a, k, array_b, k, array_c, m,
            &alpha, &beta, batch_size, alg_id);
    }
    else
    {
        ret_code = GemmExBatch_Inner_FP32(HIPBLAS_OP_N, HIPBLAS_OP_N,
            n, m, k,
            array_b, n, array_a, k, array_c, n,
            &alpha, &beta, batch_size, alg_id);
    }

    hipDeviceSynchronize();

    if (ret_code != HIPBLAS_STATUS_SUCCESS) {
        LogError("cuBLAS error %d in calling hipblasGemmEx", ret_code);
        return false;
    }
    return true;
}

bool CublasEngine::GemmEx_F32(int m, int n, int k, const float *A, const float *B, float *C,
    float alpha, float beta, bool is_b_column_major, int alg_id)
{
    hipblasStatus_t ret_code = HIPBLAS_STATUS_SUCCESS;
    if (is_b_column_major)
    {
        ret_code = GemmEx_Inner_FP32(HIPBLAS_OP_T, HIPBLAS_OP_N,
            m, n, k,
            A, k, B, k, C, m,
            &alpha, &beta, alg_id);
    }
    else
    {
        ret_code = GemmEx_Inner_FP32(HIPBLAS_OP_N, HIPBLAS_OP_N,
            n, m, k,
            B, n, A, k, C, n,
            &alpha, &beta, alg_id);
    }

    hipDeviceSynchronize();

    if (ret_code != HIPBLAS_STATUS_SUCCESS) {
        LogError("cuBLAS error %d in calling hipblasGemmEx", ret_code);
        return false;
    }
    return true;
}

bool CublasEngine::Gemv(DeviceTensor &Y, const DeviceTensor &X, const DeviceTensor &A,
    float alpha, float beta, bool is_b_column_major)
{
    bool ret = true;
    int m = A.Rows(), n = A.Columns();

    bool is_fp16 = A.data_type == ElementType::F16 && X.data_type == ElementType::F16
        && Y.data_type == ElementType::F16;
    bool is_fp32 = A.data_type == ElementType::F32 && X.data_type == ElementType::F32
        && Y.data_type == ElementType::F32;

    if (is_fp16)
    {
        const half *a_data = A.data_f16();
        const half *x_data = X.data_f16();
        half *y_data = Y.data_f16();
        //bool is_comp32 = false;
        ret = Gemv_F16(m, n, x_data, a_data, y_data, alpha, beta,
            is_b_column_major);
    }
    else if (is_fp32)
    {
        const float *a_data = A.data_f32();
        const float *x_data = X.data_f32();
        float *y_data = Y.data_f32();
        ret = Gemv_F32(m, n, x_data, a_data, y_data, alpha, beta,
            is_b_column_major);
    }
    else
    {
        ret = false;
    }

    return ret;
}

bool CublasEngine::Gemv_F16(int m, int n, const half *X, const half *A, half *Y,
    float alpha, float beta, bool is_b_column_major)
{
    (void)alpha; (void)beta; (void)m; (void)n; (void)X; (void)A; (void)Y;
    hipblasStatus_t ret_code = HIPBLAS_STATUS_SUCCESS;
    if (is_b_column_major)
    {
    }
    else
    {
        //int batch_size = 1;
        //ret_code = cublasHSHgemvBatched(handle_, HIPBLAS_OP_N, n, m,
        //    &alpha, &A, n, &X, 1, &beta, &Y, 1, batch_size);
    }

    hipDeviceSynchronize();

    if (ret_code != HIPBLAS_STATUS_SUCCESS) {
        LogError("cuBLAS error %d in calling hipblasGemmEx", ret_code);
        return false;
    }
    return true;
}

bool CublasEngine::Gemv_F32(int m, int n, const float *X, const float *A, float *Y,
    float alpha, float beta, bool is_b_column_major)
{
    hipblasStatus_t ret_code = HIPBLAS_STATUS_SUCCESS;
    if (is_b_column_major)
    {
    }
    else
    {
        ret_code = hipblasSgemv(handle_, HIPBLAS_OP_N, n, m,
            &alpha, A, n, X, 1, &beta, Y, 1);
    }

    hipDeviceSynchronize();

    if (ret_code != HIPBLAS_STATUS_SUCCESS) {
        LogError("cuBLAS error %d in calling hipblasGemmEx", ret_code);
        return false;
    }
    return true;
}

hipblasStatus_t CublasEngine::GemmEx_Inner_FP16_Comp16(hipblasOperation_t trans_a, hipblasOperation_t trans_b,
    int m, int n, int k, const half *A, int lda, const half *B, int ldb, half *C, int ldc,
    const float *alpha, const float *beta, int alg_id)
{
    hipDataType AType = HIP_R_16F;
    hipDataType BType = HIP_R_16F;
    hipDataType CType = HIP_R_16F;
    hipblasComputeType_t ComputeType = HIPBLAS_COMPUTE_32F_FAST_16F; //HIPBLAS_COMPUTE_16F;
    //hipDataType ComputeType = HIP_R_16F;
    hipblasStatus_t status = hipblasGemmEx(handle_,
        trans_a, trans_b, m, n, k,
        alpha, A, AType, lda, B, BType, ldb,
        beta, C, CType, ldc,
        ComputeType, (hipblasGemmAlgo_t)alg_id);
    return status;
}

hipblasStatus_t CublasEngine::GemmEx_Inner_FP16_Comp32(hipblasOperation_t trans_a, hipblasOperation_t trans_b,
    int m, int n, int k, const half *A, int lda, const half *B, int ldb, half *C, int ldc,
    const float *alpha, const float *beta, int alg_id)
{
    hipDataType AType = HIP_R_16F;
    hipDataType BType = HIP_R_16F;
    hipDataType CType = HIP_R_16F;
    hipblasComputeType_t ComputeType = HIPBLAS_COMPUTE_32F;
    hipblasStatus_t status = hipblasGemmEx(handle_,
        trans_a, trans_b, m, n, k,
        alpha, A, AType, lda, B, BType, ldb,
        beta, C, CType, ldc,
        ComputeType, (hipblasGemmAlgo_t)alg_id);
    return status;
}

hipblasStatus_t CublasEngine::GemmEx_Inner_FP32(hipblasOperation_t trans_a, hipblasOperation_t trans_b,
    int m, int n, int k, const float *A, int lda, const float *B, int ldb, float *C, int ldc,
    const float *alpha, const float *beta, int alg_id)
{
    hipDataType AType = HIP_R_32F;
    hipDataType BType = HIP_R_32F;
    hipDataType CType = HIP_R_32F;
    hipDataType ComputeType = HIP_R_32F;
    hipblasStatus_t status = hipblasGemmEx(handle_,
        trans_a, trans_b, m, n, k,
        alpha, A, AType, lda, B, BType, ldb,
        beta, C, CType, ldc,
        ComputeType, (hipblasGemmAlgo_t)alg_id);
    return status;
}

hipblasStatus_t CublasEngine::GemmExBatch_Inner_FP16_Comp16(hipblasOperation_t trans_a,
    hipblasOperation_t trans_b, int m, int n, int k, const half **array_a, int lda,
    const half **array_b, int ldb, half **array_c, int ldc,
    const float *alpha, const float *beta, int batch_size, int alg_id)
{
    hipDataType AType = HIP_R_16F;
    hipDataType BType = HIP_R_16F;
    hipDataType CType = HIP_R_16F;
    hipblasComputeType_t ComputeType = HIPBLAS_COMPUTE_32F_FAST_16F; //HIPBLAS_COMPUTE_16F;
    hipblasStatus_t status = hipblasGemmBatchedEx(handle_,
        trans_a, trans_b, m, n, k,
        alpha, (const void**)array_a, AType, lda,
        (const void**)array_b, BType, ldb,
        beta, (void**)array_c, CType, ldc,
        batch_size, ComputeType,
        (hipblasGemmAlgo_t)alg_id);
    return status;
}

hipblasStatus_t CublasEngine::GemmExBatch_Inner_FP16_Comp32(hipblasOperation_t trans_a,
    hipblasOperation_t trans_b, int m, int n, int k, const half **array_a, int lda,
    const half **array_b, int ldb, half **array_c, int ldc,
    const float *alpha, const float *beta, int batch_size, int alg_id)
{
    hipDataType AType = HIP_R_16F;
    hipDataType BType = HIP_R_16F;
    hipDataType CType = HIP_R_16F;
    hipblasComputeType_t ComputeType = HIPBLAS_COMPUTE_32F;
    hipblasStatus_t status = hipblasGemmBatchedEx(handle_,
        trans_a, trans_b, m, n, k,
        alpha, (const void**)array_a, AType, lda,
        (const void**)array_b, BType, ldb,
        beta, (void**)array_c, CType, ldc,
        batch_size, ComputeType,
        (hipblasGemmAlgo_t)alg_id);
    return status;
}

hipblasStatus_t CublasEngine::GemmExBatch_Inner_FP32(hipblasOperation_t trans_a,
    hipblasOperation_t trans_b, int m, int n, int k, const float **array_a, int lda,
    const float **array_b, int ldb, float **array_c, int ldc,
    const float *alpha, const float *beta, int batch_size, int alg_id)
{
    hipDataType AType = HIP_R_32F;
    hipDataType BType = HIP_R_32F;
    hipDataType CType = HIP_R_32F;
    hipDataType ComputeType = HIP_R_32F;
    hipblasStatus_t status = hipblasGemmBatchedEx(handle_,
        trans_a, trans_b, m, n, k,
        alpha, (const void**)array_a, AType, lda,
        (const void**)array_b, BType, ldb,
        beta, (void**)array_c, CType, ldc,
        batch_size, ComputeType,
        (hipblasGemmAlgo_t)alg_id);
    return status;
}

INFER_FLOW_END
